#include <stdio.h> 
#include <hip/hip_runtime.h>

using namespace std;

__global__ void cuda_function()
{
    printf("Hello World from GPU!\n");
    __syncthreads();               // to synchronize all threads
}

int main()
{
    cuda_function <<<1, 1>>> ();  // <<<block, thread>>>
    hipDeviceSynchronize();      // to synchronize device call
    return 0;
}